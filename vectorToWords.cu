#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include "postinglist.cuh"
#include "util.cuh"

using namespace thrust;

namespace postinglist {

__host__ __device__
bool isAlpha2(char c){
  return (c >= 'A' && c <= 'z');
}

struct IsAlpha : std::unary_function<char, bool>{
  __host__ __device__
  bool operator()(const char c){
	return isAlpha2(c);
  }
};

struct CharToHash : std::unary_function<char, hash_t>{
  __host__ __device__
  hash_t operator() (const char c){
	return static_cast<hash_t>(c);
  }
};

static const int WSIZE = 128;
__constant__ uint32_t hash_mult[WSIZE];

struct WordReducer {
  template<typename Tuple>
  __host__ __device__
  Tuple operator() (Tuple lhs, Tuple rhs){
	return Tuple
	  (thrust::get<0>(lhs) * hash_mult[thrust::get<2>(rhs)] + thrust::get<0>(rhs), // hash
	   thrust::get<1>(lhs), // position
	   thrust::get<2>(lhs) + thrust::get<2>(rhs)); // length
  }
};

void vectorToWords(const thrust::device_vector<char>& input,
				   thrust::device_vector<bool>& isWords,
				   thrust::device_vector<word_t>& words){
  uint64_t* hash_mult_host = new uint64_t[WSIZE];
  uint64_t hash = 33;
  for (int i = 1; i < WSIZE; i++) {
	  hash_mult_host[i] = hash;
	  hash *= 33;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(hash_mult), hash_mult_host, WSIZE*sizeof(uint64_t));

  uint32_t size = input.size();
  reduce_by_key
	(make_transform_iterator
	 (input.begin(), IsAlpha()), // key begin
	 make_transform_iterator(input.begin(), IsAlpha()) + size, // key end
	 make_zip_iterator
	 (make_tuple
	  (make_transform_iterator(input.begin(), CharToHash()), // values begin
	   counting_iterator<position_t>(0),
	   constant_iterator<length_t>(1))),
	 isWords.begin(),// reduced key begin
	 words.begin(), // reduced val begin
	 thrust::equal_to<bool>(),// key equality
	 WordReducer());// reduce op);
}
}
